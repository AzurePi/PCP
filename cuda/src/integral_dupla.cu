#include "hip/hip_runtime.h"
#include "integral_dupla.h"

#include <bits/time.h>

float cpuSecond() {
    struct timespec tp;
    clock_gettime(CLOCK_MONOTONIC, &tp);
    return (float)tp.tv_sec + tp.tv_nsec * 1.e-9f;
}

int main(int argc, char *argv[]) {
    if (argc < 4) {
        fprintf(stderr, "Uso: %s <X_INTERVALOS> <Y_INTERVALOS> <BLOCOS>\n", argv[0]);
        return 1;
    }

    // Intervalo da integral
    float limite_inf = 0.0f, limite_sup = 1.5f;

    // Parâmetros de entrada
    int x_intervalos = atoi(argv[1]); // Número de intervalos em x
    int y_intervalos = atoi(argv[2]); // Número de intervalos em y
    int n_blocos = atoi(argv[3]); // Número de blocos CUDA

    // Passo dos intervalos
    float h_x = (limite_sup - limite_inf) / x_intervalos;
    float h_y = (limite_sup - limite_inf) / y_intervalos;

    // Variável para armazenar o resultado final
    float *resultado_d, resultado_h = 0.0f;
    hipMalloc((void **)&resultado_d, sizeof(float));
    hipMemcpy(resultado_d, &resultado_h, sizeof(float), hipMemcpyHostToDevice);

    double total_time = 0.0f;

    // Realizar 10 execuções para calcular o tempo médio
    for (int i = 0; i < 10; ++i) {
        hipMemset(resultado_d, 0, sizeof(float)); // Reinicializar o resultado na memória do dispositivo

        // Medir o tempo de execução
        const double start = cpuSecond();
        integral_dupla_cuda<<<n_blocos, THREADS_POR_BLOCO>>>(resultado_d, h_x, h_y, x_intervalos, y_intervalos,
                                                             limite_inf);
        hipDeviceSynchronize();
        const double end = cpuSecond();
        const double run_time = end - start;
        total_time += run_time;

        // Recuperar o resultado
        hipMemcpy(&resultado_h, resultado_d, sizeof(float), hipMemcpyDeviceToHost);
        printf("Resultado da integral: %f | Tempo: %lf\n", resultado_h, run_time);
    }

    hipFree(resultado_d);

    resultado_h *= h_x * h_y; // Ajustar o resultado pelo tamanho dos passos dx e dy

    double avg_time = total_time / 10.0;

    printf("Tempo médio com %d bloco(s), X=%d, Y=%d: %f segundos\n", n_blocos, x_intervalos, y_intervalos, avg_time);
    salvar_tempos(n_blocos, x_intervalos, y_intervalos, avg_time);

    return 0;
}

// Kernel CUDA para calcular a integral dupla
__global__ void integral_dupla_cuda(float *resultado, float h_x, float h_d, int x_intervalos, int y_intervalos,
                                    float limite_inf) {
    unsigned const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned const int totalThreads = gridDim.x * blockDim.x;

    float soma = 0.0f;

    // Cada thread processa múltiplos intervalos
    for (unsigned int i = idx; i < x_intervalos * y_intervalos; i += totalThreads) {
        unsigned const int ix = i % x_intervalos;
        unsigned const int iy = i / x_intervalos;

        float const x = limite_inf + ix * h_x;
        float const y = limite_inf + iy * h_d;

        float const f = F(x, y);
        if (ix == 0 || ix == x_intervalos - 1 || iy == 0 || iy == y_intervalos - 1)
            soma += f * 0.5f; // borda
        else
            soma += f; // interior
    }

    // Soma parcial da thread
    atomicAdd(resultado, soma);
}

void salvar_tempos(int blocos, int x_intervalos, int y_intervalos, double tempo_medio) {
    const char *output_file = "tempos_CUDA.txt";

    // Verifica se o arquivo já existe
    int arquivo_existe = access(output_file, F_OK) == 0;

    FILE *resultados = fopen(output_file, "a");
    if (resultados == NULL) {
        perror("Erro ao abrir o arquivo para salvar resultados");
        return;
    }

    // Escreve o cabeçalho apenas na primeira vez
    if (!arquivo_existe) {
        fprintf(resultados, "Blocos\tIntervalo_X\tIntervalo_Y\tTempo_Medio(s)\n");
        fprintf(resultados, "---------------------------------------------\n");
    }

    // Salva os dados no arquivo
    fprintf(resultados, "%d\t\t%d\t\t%d\t\t%lf\n", blocos, x_intervalos, y_intervalos, tempo_medio);

    fclose(resultados);
    printf("Resultados salvos em %s\n", output_file);
}
